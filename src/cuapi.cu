#include "cuapi.h"

// Specializations for float
hipblasStatus_t cublasAPI<float>::I_amax(hipblasHandle_t handle, int n, const float* x, int incx, int* result) {
    return hipblasIsamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasAPI<float>::Axpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy) {
    return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasAPI<float>::Scal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx) {
    return hipblasSscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublasAPI<float>::Gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float* alpha,
                                     const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc) {
    return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

// Specializations for double
hipblasStatus_t cublasAPI<double>::I_amax(hipblasHandle_t handle, int n, const double* x, int incx, int* result) {
    return hipblasIdamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasAPI<double>::Axpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy) {
    return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasAPI<double>::Scal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx) {
    return hipblasDscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublasAPI<double>::Gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const double* alpha,
                                     const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc) {
    return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

// Specializations for hipComplex
hipblasStatus_t cublasAPI<hipComplex>::I_amax(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result) {
    return hipblasIcamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasAPI<hipComplex>::Axpy(hipblasHandle_t handle, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* y, int incy) {
    return hipblasCaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasAPI<hipComplex>::Scal(hipblasHandle_t handle, int n, const hipComplex* alpha, hipComplex* x, int incx) {
    return hipblasCscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublasAPI<hipComplex>::Gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const hipComplex* alpha,
                                         const hipComplex* A, int lda, const hipComplex* B, int ldb, const hipComplex* beta, hipComplex* C, int ldc) {
    return hipblasCgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

// Specializations for hipDoubleComplex
hipblasStatus_t cublasAPI<hipDoubleComplex>::I_amax(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result) {
    return hipblasIzamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasAPI<hipDoubleComplex>::Axpy(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy) {
    return hipblasZaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasAPI<hipDoubleComplex>::Scal(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int incx) {
    return hipblasZscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublasAPI<hipDoubleComplex>::Gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const hipDoubleComplex* alpha,
                                               const hipDoubleComplex* A, int lda, const hipDoubleComplex* B, int ldb, const hipDoubleComplex* beta, hipDoubleComplex* C, int ldc) {
    return hipblasZgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

// Specializations for cusolverAPI functions
void cusolverAPI<float>::Dn_getrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* Lwork) {
    hipsolverDnSgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

void cusolverAPI<float>::Dn_getrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* Workspace, int* devIpiv, int* devInfo) {
    hipsolverDnSgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

void cusolverAPI<float>::Dn_getrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo) {
    hipsolverDnSgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}



// Add similar specializations for double, hipComplex, and hipDoubleComplex

void cusolverAPI<double>::Dn_getrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* Lwork) {
    hipsolverDnDgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

void cusolverAPI<double>::Dn_getrf(hipsolverHandle_t handle, int m, int n, double* A, int lda, double* Workspace, int* devIpiv, int* devInfo) {
    hipsolverDnDgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

void cusolverAPI<double>::Dn_getrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo) {
    hipsolverDnDgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}

void cusolverAPI<hipComplex>::Dn_getrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipComplex* A, int lda, int* Lwork) {
    hipsolverDnCgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

void cusolverAPI<hipComplex>::Dn_getrf(hipsolverHandle_t handle, int m, int n, hipComplex* A, int lda, hipComplex* Workspace, int* devIpiv, int* devInfo) {
    hipsolverDnCgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

void cusolverAPI<hipComplex>::Dn_getrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipComplex* A, int lda, const int* devIpiv, hipComplex* B, int ldb, int* devInfo) {
    hipsolverDnCgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}


void cusolverAPI<hipDoubleComplex>::Dn_getrf_bufferSize(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, int* Lwork) {
    hipsolverDnZgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

void cusolverAPI<hipDoubleComplex>::Dn_getrf(hipsolverHandle_t handle, int m, int n, hipDoubleComplex* A, int lda, hipDoubleComplex* Workspace, int* devIpiv, int* devInfo) {
    hipsolverDnZgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

void cusolverAPI<hipDoubleComplex>::Dn_getrs(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs, const hipDoubleComplex* A, int lda, const int* devIpiv, hipDoubleComplex* B, int ldb, int* devInfo) {
    hipsolverDnZgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}