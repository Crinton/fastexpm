#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_complex.h>
#include <time.h>
#include <cmath>
#include <utility>
#include <type_traits>
#include "expm.h"
#include "util.h"
#include "cuapi.h"
void checkCudaError(hipError_t error) {
    if (error == hipSuccess) {
    } else {
        const char *errorStr = hipGetErrorString(error);
        printf("CUDA Error: %s\n", errorStr);
    }
}

void checkCublasStatus(hipblasStatus_t status) {
    if (status == HIPBLAS_STATUS_SUCCESS) {

    } else {
        switch (status) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                printf("CUBLAS library not initialized\n");
                break;
            case HIPBLAS_STATUS_ALLOC_FAILED:
                printf("Resource allocation failed\n");
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                printf("An invalid value was used as an argument\n");
                break;
            case HIPBLAS_STATUS_ARCH_MISMATCH:
                printf("An unsupported CUDA architecture was used\n");
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                printf("An access to GPU memory space failed\n");
                break;
            case HIPBLAS_STATUS_EXECUTION_FAILED:
                printf("The GPU program failed to execute\n");
                break;
            case HIPBLAS_STATUS_INTERNAL_ERROR:
                printf("An internal operation failed\n");
                break;
            default:
                printf("An unknown error occurred\n");
                break;
        }
    }
}

// template <typename T>
// double trace(T *A,const int M)
// {
//     T tr = 0;
//     for(int i=0; i<M; ++i)
//     {
//         tr += A[i*M + i];
//     }
//     return tr;
// }

void CuComplexToStdComplex_Array(hipComplex *A, std::complex<float> *B, int M)
{
    hipMemcpy(B,A,M*M*sizeof(hipComplex),hipMemcpyDeviceToHost);
    for (int i=0;i < M*M; ++i)
    {
        B[i] = std::complex<float>(A[i].x, A[i].y);
    }
}

void CuComplexToStdComplex_Array(hipDoubleComplex *A, std::complex<double> *B, int M)
{
    hipMemcpy(B,A,M*M*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
    for (int i=0;i < M*M; ++i)
    {
        B[i] = std::complex<double>(A[i].x, A[i].y);
    }
}

template <typename T>
hipComplex TocuComplex(T x){
    return make_hipComplex(x,0.0f);
}

template <typename T>
hipDoubleComplex TocuDoubleComplex(T x){
    return make_hipDoubleComplex(x,0.0f);
}

static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
	double factor = exp(x.x);
	return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}

static __inline__ hipComplex cuCexp(hipComplex x)
{
	double factor = exp(x.x);
	return make_hipComplex(factor * cos(x.y), factor * sin(x.y));
}


std::pair<float, int> pre_processing(float *A, float *d_A, int M) //预处理的结果是d_A
{
    double theta = 5.371920351148152;
    size_t matrixCount = M*M*sizeof(float);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    checkCudaError(hipMemcpy(d_A, A, matrixCount, hipMemcpyHostToDevice));
    float A_trace,mu,*d_I;
    float L1_norm;
    hipMalloc((void**)&d_I, M*M*sizeof(float));
    A_trace = trace(A,M); // A_trace 是 std::complex
    mu = - A_trace / (float)M; // mu = - trace(A) / M  
    eye(d_I,M);
    // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    // mu需要转化一下类型
    checkCublasStatus(cublasAPI<float>::Axpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    checkCudaError(hipFree(d_I));
    L1_norm = matrix_L1_norm(d_A,M); // 得到转化后的A的第一范数
    hipDeviceSynchronize();
    int s = (int)std::ceil(std::log2(L1_norm/theta));
    float scala = 1.0f / std::pow(2,s);

    // checkCublasStatus(hipblasDscal(cublasH,M*M,&scala,d_A,1)); //  A = (A / (2**s))
    checkCublasStatus(cublasAPI<float>::Scal(cublasH,M*M,&scala,d_A,1));
    hipblasDestroy(cublasH);
    return std::make_pair(mu,s);
}

std::pair<double, int> pre_processing(double *A, double *d_A, int M) //预处理的结果是d_A
{
    double theta = 5.371920351148152;
    size_t matrixCount = M*M*sizeof(double);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    checkCudaError(hipMemcpy(d_A, A, matrixCount, hipMemcpyHostToDevice));
    double A_trace,mu,*d_I;
    double L1_norm;
    hipMalloc((void**)&d_I, M*M*sizeof(double));
    A_trace = trace(A,M); // A_trace 是 std::complex
    mu = - A_trace / (double)M; // mu = - trace(A) / M  
    eye(d_I,M);
    // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    // mu需要转化一下类型
    checkCublasStatus(cublasAPI<double>::Axpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    checkCudaError(hipFree(d_I));
    L1_norm = matrix_L1_norm(d_A,M); // 得到转化后的A的第一范数
    hipDeviceSynchronize();
    int s = (int)std::ceil(std::log2(L1_norm/theta));
    double scala = 1.0f / std::pow(2,s);

    // checkCublasStatus(hipblasDscal(cublasH,M*M,&scala,d_A,1)); //  A = (A / (2**s))
    checkCublasStatus(cublasAPI<double>::Scal(cublasH,M*M,&scala,d_A,1));
    hipblasDestroy(cublasH);
    return std::make_pair(mu,s);
}

std::pair<hipComplex, int> pre_processing(std::complex<float> *A, hipComplex *d_A, int M) //预处理的结果是d_A
{
    double theta = 5.371920351148152;
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    checkCudaError(hipMemcpy(d_A, A, M*M*sizeof(hipComplex), hipMemcpyHostToDevice));
    std::complex<float> A_trace,mu_host;
    hipComplex *d_I;
    double L1_norm;
    hipMalloc((void**)&d_I, M*M*sizeof(hipComplex));
    A_trace = trace(A,M); // A_trace 是 std::complex
    mu_host = - A_trace / std::complex<float>((float)M,0.0f); // mu = - trace(A) / M 
    hipComplex mu = make_hipComplex(mu_host.real(),mu_host.imag());
    eye(d_I,M);
    // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    // mu需要转化一下类型
    checkCublasStatus(cublasAPI<hipComplex>::Axpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    checkCudaError(hipFree(d_I));
    L1_norm = matrix_L1_norm(d_A,M); // 得到转化后的A的第一范数
    hipDeviceSynchronize();
    int s = (int)std::ceil(std::log2(L1_norm/theta));
    double scala_tmp = 1.0f / std::pow(2,s);
    hipComplex scala = make_hipComplex(scala_tmp,0.0f);
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&scala,d_A,1)); //  A = (A / (2**s))
    checkCublasStatus(cublasAPI<hipComplex>::Scal(cublasH,M*M,&scala,d_A,1));
    hipblasDestroy(cublasH);
    return std::make_pair(mu,s);
}

std::pair<hipDoubleComplex, int> pre_processing(std::complex<double> *A, hipDoubleComplex *d_A, int M) //预处理的结果是d_A
{
    double theta = 5.371920351148152;
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    checkCudaError(hipMemcpy(d_A, A, M*M*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    std::complex<double> A_trace,mu_host;
    hipDoubleComplex *d_I;
    double L1_norm;
    hipMalloc((void**)&d_I, M*M*sizeof(hipDoubleComplex));
    A_trace = trace(A,M); // A_trace 是 std::complex
    mu_host = - A_trace / std::complex<double>((double)M,0.0f); // mu = - trace(A) / M 
    hipDoubleComplex mu = make_hipDoubleComplex(mu_host.real(),mu_host.imag());
    eye(d_I,M);
    // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    // mu需要转化一下类型
    checkCublasStatus(cublasAPI<hipDoubleComplex>::Axpy(cublasH, M*M, &mu, d_I, 1, d_A, 1));
    checkCudaError(hipFree(d_I));
    L1_norm = matrix_L1_norm(d_A,M); // 得到转化后的A的第一范数
    hipDeviceSynchronize();
    int s = (int)std::ceil(std::log2(L1_norm/theta));
    double scala_tmp = 1.0f / std::pow(2,s);
    hipDoubleComplex scala = make_hipDoubleComplex(scala_tmp,0.0f);
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&scala,d_A,1)); //  A = (A / (2**s))
    checkCublasStatus(cublasAPI<hipDoubleComplex>::Scal(cublasH,M*M,&scala,d_A,1));
    hipblasDestroy(cublasH);
    return std::make_pair(mu,s);
}


float* pade_appromixmate(float *d_A, int M)
{   
    int m = 14;
    size_t matrixCount = M*M*sizeof(float);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;;
    hipsolverDnCreate(&cusolverH);

    checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    hipsolverSetStream(cusolverH, stream);
    hipblasSetStream(cublasH,stream);
    float poly[m] = {1,0.5,0.11999999731779099,0.018333332613110542,0.0019927537068724632,0.00016304348537232727,
                                1.035196692100726e-05, 5.175983233129955e-07, 2.0431513566525008e-08,6.306022705717595e-10,1.48377004840414e-11,2.529153491597966e-13,
                                2.8101705462199623e-15,1.5440497506703088e-17};
    float *d_matrix_tmp, *d_P, *d_Q,*ggtmp;
    hipMallocAsync((void**)&d_matrix_tmp, matrixCount,stream);
    hipMallocAsync((void**)&d_P, matrixCount,stream);
    hipMallocAsync((void**)&d_Q, matrixCount,stream);
    hipMallocAsync((void**)&ggtmp, matrixCount,stream);
    eye(d_matrix_tmp,M);
    //  d_P <- I; d_Q <- I
    hipStreamSynchronize(stream);
    checkCudaError(hipMemcpyAsync(d_P, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    checkCudaError(hipMemcpyAsync(d_Q, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    hipStreamSynchronize(stream);
    //hipMemcpy(d_B, B, M*M * sizeof(float), cudaMemc"yHostToDevice);
    //这一步是直接覆盖的 result_P = poly[0] * torch.eye(A.shape[0],dtype=A.dtype).cuda()
    checkCublasStatus(cublasAPI<float>::Scal(cublasH,M*M,&poly[0],d_P,1));
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&poly[0],d_Q,1));
    float alpha=1.0f,beta=0.0f;
    float alpha_add;
    for(int i=1;i<m;++i)
    {
        //matrix_tmp = matrix_tmp @ A 
        hipStreamSynchronize(stream);
        // hipblasDgemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M);
        checkCublasStatus(cublasAPI<float>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M));
        hipStreamSynchronize(stream);
        checkCudaError(hipMemcpyAsync(d_matrix_tmp,ggtmp,matrixCount,hipMemcpyDeviceToDevice,stream));
        // result_P = result_P + poly[i] * matrix_tmp
        // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1)); // 这里利用向量加法实现矩阵加法
        checkCublasStatus(cublasAPI<float>::Axpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1));
        //result_Q = result_Q + ((-1)**i) * poly[i]* matrix_tmp
        alpha_add = ((i % 2 == 0) ? 1.0 : -1.0) * poly[i];
        checkCublasStatus(cublasAPI<float>::Axpy(cublasH, M*M, &alpha_add, ggtmp, 1, d_Q, 1)); 
    }
    checkCudaError(hipFreeAsync(d_A,stream));
    checkCudaError(hipFreeAsync(d_matrix_tmp,stream));
    checkCudaError(hipFreeAsync(ggtmp,stream));
    checkCublasStatus(hipblasDestroy(cublasH));
    int lda = M;
    int ldb = M;

    /*
    以下参数为LU分解求解Linear System的参数设置
    */
    int info = 0;
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */
    int lwork = 0;            /* size of workspace */
    float *d_work = nullptr; /* device workspace for getrf */
    // const int pivot_on = 1;
    // if (pivot_on)
    // {
    //     printf("pivot is on : compute P*A = L*U \n");
    // }
    // else
    // {
    //     printf("pivot is off: compute A = L*U (not numerically stable)\n");
    // } 
    /* step 2: copy A to device */
    checkCudaError(hipMallocAsync((void **)(&d_Ipiv), sizeof(int) * M,stream));
    checkCudaError(hipMallocAsync((void **)(&d_info), sizeof(int),stream));
    /* step 3: query working space of getrf */
    // (hipsolverDnDgetrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork));
    cusolverAPI<float>::Dn_getrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork);
    checkCudaError(hipMalloc((void **)(&d_work), sizeof(float) * lwork));
    /* step 4: LU factorization */

    // (hipsolverDnDgetrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info));
    cusolverAPI<float>::Dn_getrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info);

    (hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    (hipStreamSynchronize(stream));
    // if (0 > info)
    // {
    //     printf("%d-th parameter is wrong \n", -info);
    //     exit(1);
    // }
    // if (pivot_on)
    // {
    //     printf("pivoting sequence, matlab base-1\n");
    // }
    /*
     * step 5: solve A*X = B
     */
    int * Ipiv = (int *)malloc(M*sizeof(int));
    hipMemcpy(Ipiv,d_Ipiv,M*sizeof(int),hipMemcpyDeviceToHost);

    // (hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info));
    cusolverAPI<float>::Dn_getrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info);
    (hipStreamSynchronize(stream));
    /* free resources */
    hipDeviceSynchronize();
    checkCudaError(hipFree(d_Q));
    checkCudaError(hipFree(d_Ipiv));
    checkCudaError(hipFree(d_info));
    checkCudaError(hipFree(d_work));
    (hipsolverDnDestroy(cusolverH));
    return d_P;
}

double* pade_appromixmate(double *d_A, int M)
{   
    int m = 14;
    size_t matrixCount = M*M*sizeof(double);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;;
    hipsolverDnCreate(&cusolverH);

    checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    hipsolverSetStream(cusolverH, stream);
    hipblasSetStream(cublasH,stream);
    double poly[m] = {1,0.5,0.11999999731779099,0.018333332613110542,0.0019927537068724632,0.00016304348537232727,
                                1.035196692100726e-05, 5.175983233129955e-07, 2.0431513566525008e-08,6.306022705717595e-10,1.48377004840414e-11,2.529153491597966e-13,
                                2.8101705462199623e-15,1.5440497506703088e-17};
    double *d_matrix_tmp, *d_P, *d_Q,*ggtmp;
    hipMallocAsync((void**)&d_matrix_tmp, matrixCount,stream);
    hipMallocAsync((void**)&d_P, matrixCount,stream);
    hipMallocAsync((void**)&d_Q, matrixCount,stream);
    hipMallocAsync((void**)&ggtmp, matrixCount,stream);
    eye(d_matrix_tmp,M);
    //  d_P <- I; d_Q <- I
    hipStreamSynchronize(stream);
    checkCudaError(hipMemcpyAsync(d_P, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    checkCudaError(hipMemcpyAsync(d_Q, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    hipStreamSynchronize(stream);
    //hipMemcpy(d_B, B, M*M * sizeof(float), cudaMemc"yHostToDevice);
    //这一步是直接覆盖的 result_P = poly[0] * torch.eye(A.shape[0],dtype=A.dtype).cuda()
    checkCublasStatus(cublasAPI<double>::Scal(cublasH,M*M,&poly[0],d_P,1));
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&poly[0],d_Q,1));
    double alpha=1.0f,beta=0.0f;
    double alpha_add;
    for(int i=1;i<m;++i)
    {
        //matrix_tmp = matrix_tmp @ A 
        hipStreamSynchronize(stream);
        // hipblasDgemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M);
        checkCublasStatus(cublasAPI<double>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M));
        hipStreamSynchronize(stream);
        checkCudaError(hipMemcpyAsync(d_matrix_tmp,ggtmp,matrixCount,hipMemcpyDeviceToDevice,stream));
        // result_P = result_P + poly[i] * matrix_tmp
        // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1)); // 这里利用向量加法实现矩阵加法
        checkCublasStatus(cublasAPI<double>::Axpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1));
        //result_Q = result_Q + ((-1)**i) * poly[i]* matrix_tmp
        alpha_add = ((i % 2 == 0) ? 1.0 : -1.0) * poly[i];
        checkCublasStatus(cublasAPI<double>::Axpy(cublasH, M*M, &alpha_add, ggtmp, 1, d_Q, 1)); 
    }
    checkCudaError(hipFreeAsync(d_A,stream));
    checkCudaError(hipFreeAsync(d_matrix_tmp,stream));
    checkCudaError(hipFreeAsync(ggtmp,stream));
    checkCublasStatus(hipblasDestroy(cublasH));
    int lda = M;
    int ldb = M;

    /*
    以下参数为LU分解求解Linear System的参数设置
    */
    int info = 0;
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */
    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */
    // const int pivot_on = 1;
    // if (pivot_on)
    // {
    //     printf("pivot is on : compute P*A = L*U \n");
    // }
    // else
    // {
    //     printf("pivot is off: compute A = L*U (not numerically stable)\n");
    // } 
    /* step 2: copy A to device */
    checkCudaError(hipMallocAsync((void **)(&d_Ipiv), sizeof(int) * M,stream));
    checkCudaError(hipMallocAsync((void **)(&d_info), sizeof(int),stream));
    /* step 3: query working space of getrf */
    // (hipsolverDnDgetrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork));
    cusolverAPI<double>::Dn_getrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork);
    checkCudaError(hipMalloc((void **)(&d_work), sizeof(double) * lwork));
    /* step 4: LU factorization */

    // (hipsolverDnDgetrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info));
    cusolverAPI<double>::Dn_getrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info);

    (hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    (hipStreamSynchronize(stream));
    // if (0 > info)
    // {
    //     printf("%d-th parameter is wrong \n", -info);
    //     exit(1);
    // }
    // if (pivot_on)
    // {
    //     printf("pivoting sequence, matlab base-1\n");
    // }
    /*
     * step 5: solve A*X = B
     */
    int * Ipiv = (int *)malloc(M*sizeof(int));
    hipMemcpy(Ipiv,d_Ipiv,M*sizeof(int),hipMemcpyDeviceToHost);

    // (hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info));
    cusolverAPI<double>::Dn_getrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info);
    (hipStreamSynchronize(stream));
    /* free resources */
    hipDeviceSynchronize();
    checkCudaError(hipFree(d_Q));
    checkCudaError(hipFree(d_Ipiv));
    checkCudaError(hipFree(d_info));
    checkCudaError(hipFree(d_work));
    (hipsolverDnDestroy(cusolverH));
    return d_P;
}

hipComplex* pade_appromixmate(hipComplex *d_A, int M)
{   
    int m = 14;
    size_t matrixCount = M*M*sizeof(hipComplex);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;;
    hipsolverDnCreate(&cusolverH);

    checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    hipsolverSetStream(cusolverH, stream);
    hipblasSetStream(cublasH,stream);
    double poly_tmp[m] = {1,0.5,0.11999999731779099,0.018333332613110542,0.0019927537068724632,0.00016304348537232727,
                                1.035196692100726e-05, 5.175983233129955e-07, 2.0431513566525008e-08,6.306022705717595e-10,1.48377004840414e-11,2.529153491597966e-13,
                                2.8101705462199623e-15,1.5440497506703088e-17};
    hipComplex poly[m];
    for (int i=0; i<m; ++i){
        poly[i] = TocuComplex(poly_tmp[i]);
    }

    hipComplex *d_matrix_tmp, *d_P, *d_Q,*ggtmp;
    hipMallocAsync((void**)&d_matrix_tmp, matrixCount,stream);
    hipMallocAsync((void**)&d_P, matrixCount,stream);
    hipMallocAsync((void**)&d_Q, matrixCount,stream);
    hipMallocAsync((void**)&ggtmp, matrixCount,stream);
    eye(d_matrix_tmp,M);
    //  d_P <- I; d_Q <- I
    hipStreamSynchronize(stream);
    checkCudaError(hipMemcpyAsync(d_P, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    checkCudaError(hipMemcpyAsync(d_Q, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    hipStreamSynchronize(stream);
    //hipMemcpy(d_B, B, M*M * sizeof(float), cudaMemc"yHostToDevice);
    //这一步是直接覆盖的 result_P = poly[0] * torch.eye(A.shape[0],dtype=A.dtype).cuda()
    checkCublasStatus(cublasAPI<hipComplex>::Scal(cublasH,M*M,&poly[0],d_P,1));
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&poly[0],d_Q,1));
    hipComplex alpha= TocuComplex(1.0f),beta=TocuComplex(0.0f);
    hipComplex alpha_add;
    for(int i=1;i<m;++i)
    {
        //matrix_tmp = matrix_tmp @ A 
        hipStreamSynchronize(stream);
        // hipblasDgemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M);
        checkCublasStatus(cublasAPI<hipComplex>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M));
        hipStreamSynchronize(stream);
        checkCudaError(hipMemcpyAsync(d_matrix_tmp,ggtmp,matrixCount,hipMemcpyDeviceToDevice,stream));
        // result_P = result_P + poly[i] * matrix_tmp
        // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1)); // 这里利用向量加法实现矩阵加法
        checkCublasStatus(cublasAPI<hipComplex>::Axpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1));
        //result_Q = result_Q + ((-1)**i) * poly[i]* matrix_tmp
        alpha_add = hipCmulf(TocuComplex((i % 2 == 0) ? 1.0 : -1.0), poly[i]);
        checkCublasStatus(cublasAPI<hipComplex>::Axpy(cublasH, M*M, &alpha_add, ggtmp, 1, d_Q, 1)); 
    }
    checkCudaError(hipFreeAsync(d_A,stream));
    checkCudaError(hipFreeAsync(d_matrix_tmp,stream));
    checkCudaError(hipFreeAsync(ggtmp,stream));
    checkCublasStatus(hipblasDestroy(cublasH));
    int lda = M;
    int ldb = M;

    /*
    以下参数为LU分解求解Linear System的参数设置
    */
    int info = 0;
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */
    int lwork = 0;            /* size of workspace */
    hipComplex *d_work = nullptr; /* device workspace for getrf */
    // const int pivot_on = 1;
    // if (pivot_on)
    // {
    //     printf("pivot is on : compute P*A = L*U \n");
    // }
    // else
    // {
    //     printf("pivot is off: compute A = L*U (not numerically stable)\n");
    // } 
    /* step 2: copy A to device */
    checkCudaError(hipMallocAsync((void **)(&d_Ipiv), sizeof(int) * M,stream));
    checkCudaError(hipMallocAsync((void **)(&d_info), sizeof(int),stream));
    /* step 3: query working space of getrf */
    // (hipsolverDnDgetrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork));
    cusolverAPI<hipComplex>::Dn_getrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork);
    checkCudaError(hipMalloc((void **)(&d_work), sizeof(hipComplex) * lwork));
    /* step 4: LU factorization */

    // (hipsolverDnDgetrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info));
    cusolverAPI<hipComplex>::Dn_getrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info);

    (hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    (hipStreamSynchronize(stream));
    // if (0 > info)
    // {
    //     printf("%d-th parameter is wrong \n", -info);
    //     exit(1);
    // }
    // if (pivot_on)
    // {
    //     printf("pivoting sequence, matlab base-1\n");
    // }
    /*
     * step 5: solve A*X = B
     */
    int * Ipiv = (int *)malloc(M*sizeof(int));
    hipMemcpy(Ipiv,d_Ipiv,M*sizeof(int),hipMemcpyDeviceToHost);

    // (hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info));
    cusolverAPI<hipComplex>::Dn_getrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info);
    (hipStreamSynchronize(stream));
    /* free resources */
    hipDeviceSynchronize();
    checkCudaError(hipFree(d_Q));
    checkCudaError(hipFree(d_Ipiv));
    checkCudaError(hipFree(d_info));
    checkCudaError(hipFree(d_work));
    (hipsolverDnDestroy(cusolverH));
    return d_P;
}

hipDoubleComplex* pade_appromixmate(hipDoubleComplex *d_A, int M)
{   
    int m = 14;
    size_t matrixCount = M*M*sizeof(hipDoubleComplex);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;;
    hipsolverDnCreate(&cusolverH);

    checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    hipsolverSetStream(cusolverH, stream);
    hipblasSetStream(cublasH,stream);
    double poly_tmp[m] = {1,0.5,0.11999999731779099,0.018333332613110542,0.0019927537068724632,0.00016304348537232727,
                                1.035196692100726e-05, 5.175983233129955e-07, 2.0431513566525008e-08,6.306022705717595e-10,1.48377004840414e-11,2.529153491597966e-13,
                                2.8101705462199623e-15,1.5440497506703088e-17};
    hipDoubleComplex poly[m];
    for (int i=0; i<m; ++i){
        poly[i] = TocuDoubleComplex(poly_tmp[i]);
    }

    hipDoubleComplex *d_matrix_tmp, *d_P, *d_Q,*ggtmp;
    hipMallocAsync((void**)&d_matrix_tmp, matrixCount,stream);
    hipMallocAsync((void**)&d_P, matrixCount,stream);
    hipMallocAsync((void**)&d_Q, matrixCount,stream);
    hipMallocAsync((void**)&ggtmp, matrixCount,stream);
    eye(d_matrix_tmp,M);
    //  d_P <- I; d_Q <- I
    hipStreamSynchronize(stream);
    checkCudaError(hipMemcpyAsync(d_P, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    checkCudaError(hipMemcpyAsync(d_Q, d_matrix_tmp, matrixCount, hipMemcpyDeviceToDevice,stream));
    hipStreamSynchronize(stream);
    //hipMemcpy(d_B, B, M*M * sizeof(float), cudaMemc"yHostToDevice);
    //这一步是直接覆盖的 result_P = poly[0] * torch.eye(A.shape[0],dtype=A.dtype).cuda()
    checkCublasStatus(cublasAPI<hipDoubleComplex>::Scal(cublasH,M*M,&poly[0],d_P,1));
    // checkCublasStatus(hipblasDscal(cublasH,M*M,&poly[0],d_Q,1));
    hipDoubleComplex alpha= TocuDoubleComplex(1.0f),beta=TocuDoubleComplex(0.0f);
    hipDoubleComplex alpha_add;
    for(int i=1;i<m;++i)
    {
        //matrix_tmp = matrix_tmp @ A 
        hipStreamSynchronize(stream);
        // hipblasDgemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M);
        checkCublasStatus(cublasAPI<hipDoubleComplex>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_matrix_tmp,M,d_A,M,&beta,ggtmp,M));
        hipStreamSynchronize(stream);
        checkCudaError(hipMemcpyAsync(d_matrix_tmp,ggtmp,matrixCount,hipMemcpyDeviceToDevice,stream));
        // result_P = result_P + poly[i] * matrix_tmp
        // checkCublasStatus(hipblasDaxpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1)); // 这里利用向量加法实现矩阵加法
        checkCublasStatus(cublasAPI<hipDoubleComplex>::Axpy(cublasH, M*M, &poly[i], ggtmp, 1, d_P, 1));
        //result_Q = result_Q + ((-1)**i) * poly[i]* matrix_tmp
        alpha_add = hipCmul(TocuDoubleComplex((i % 2 == 0) ? 1.0 : -1.0), poly[i]);
        checkCublasStatus(cublasAPI<hipDoubleComplex>::Axpy(cublasH, M*M, &alpha_add, ggtmp, 1, d_Q, 1)); 
    }
    checkCudaError(hipFreeAsync(d_A,stream));
    checkCudaError(hipFreeAsync(d_matrix_tmp,stream));
    checkCudaError(hipFreeAsync(ggtmp,stream));
    checkCublasStatus(hipblasDestroy(cublasH));
    int lda = M;
    int ldb = M;

    /*
    以下参数为LU分解求解Linear System的参数设置
    */
    int info = 0;
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */
    int lwork = 0;            /* size of workspace */
    hipDoubleComplex *d_work = nullptr; /* device workspace for getrf */
    // const int pivot_on = 1;
    // if (pivot_on)
    // {
    //     printf("pivot is on : compute P*A = L*U \n");
    // }
    // else
    // {
    //     printf("pivot is off: compute A = L*U (not numerically stable)\n");
    // } 
    /* step 2: copy A to device */
    checkCudaError(hipMallocAsync((void **)(&d_Ipiv), sizeof(int) * M,stream));
    checkCudaError(hipMallocAsync((void **)(&d_info), sizeof(int),stream));
    /* step 3: query working space of getrf */
    // (hipsolverDnDgetrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork));
    cusolverAPI<hipDoubleComplex>::Dn_getrf_bufferSize(cusolverH, M, M, d_Q, lda, &lwork);
    checkCudaError(hipMalloc((void **)(&d_work), sizeof(hipDoubleComplex) * lwork));
    /* step 4: LU factorization */

    // (hipsolverDnDgetrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info));
    cusolverAPI<hipDoubleComplex>::Dn_getrf(cusolverH, M, M, d_Q, lda, d_work, d_Ipiv, d_info);

    (hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    (hipStreamSynchronize(stream));
    // if (0 > info)
    // {
    //     printf("%d-th parameter is wrong \n", -info);
    //     exit(1);
    // }
    // if (pivot_on)
    // {
    //     printf("pivoting sequence, matlab base-1\n");
    // }
    /*
     * step 5: solve A*X = B
     */
    int * Ipiv = (int *)malloc(M*sizeof(int));
    hipMemcpy(Ipiv,d_Ipiv,M*sizeof(int),hipMemcpyDeviceToHost);

    // (hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info));
    cusolverAPI<hipDoubleComplex>::Dn_getrs(cusolverH, HIPBLAS_OP_N, M, M, d_Q, lda, d_Ipiv, d_P, ldb, d_info);
    (hipStreamSynchronize(stream));
    /* free resources */
    hipDeviceSynchronize();
    checkCudaError(hipFree(d_Q));
    checkCudaError(hipFree(d_Ipiv));
    checkCudaError(hipFree(d_info));
    checkCudaError(hipFree(d_work));
    (hipsolverDnDestroy(cusolverH));
    return d_P;
}


void undo_preprocessing(float *d_P, float *B, int M ,int s, float mu)
{
    float alpha = 1.0f; float beta = 0.0f; //scala = 1/ scala;
    size_t matrixCount = M*M*sizeof(float);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    float *ggtmp;
    hipMalloc((void**)&ggtmp, matrixCount);
    

    for(int i=0;i<s;++i)
    {
        //  matrix_tmp = matrix_tmp @ A 
        // hipblasDgemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_P,M,d_P,M,&beta,ggtmp,M);
        cublasAPI<float>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_P,M,d_P,M,&beta,ggtmp,M);
        checkCudaError(hipMemcpy(d_P,ggtmp,matrixCount,hipMemcpyDeviceToDevice));
    }
    float alpha_add = std::exp(mu);
    // checkCublasStatus(hipblasDscal(cublasH, M*M, &alpha_add,d_P,1)); 
    checkCublasStatus(cublasAPI<float>::Scal(cublasH, M*M, &alpha_add,d_P,1));
    checkCudaError(hipFree(ggtmp));
    checkCudaError(hipMemcpy(B,d_P,matrixCount,hipMemcpyDeviceToHost));
    checkCublasStatus(hipblasDestroy(cublasH));
}

void undo_preprocessing(double *d_P,double *B, int M ,int s, double mu)
{
    double alpha = 1.0f; double beta = 0.0f; //scala = 1/ scala;
    size_t matrixCount = M*M*sizeof(double);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    double *ggtmp;
    hipMalloc((void**)&ggtmp, matrixCount);
    

    for(int i=0;i<s;++i)
    {
        //  matrix_tmp = matrix_tmp @ A 
        cublasAPI<double>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_P,M,d_P,M,&beta,ggtmp,M);
        checkCudaError(hipMemcpy(d_P,ggtmp,matrixCount,hipMemcpyDeviceToDevice));
    }
    double alpha_add = std::exp(mu);
    checkCublasStatus(cublasAPI<double>::Scal(cublasH, M*M, &alpha_add,d_P,1));
    checkCudaError(hipFree(ggtmp));
    checkCudaError(hipMemcpy(B,d_P,matrixCount,hipMemcpyDeviceToHost));
    checkCublasStatus(hipblasDestroy(cublasH));
}

void undo_preprocessing(hipComplex *d_P,hipComplex *B, int M ,int s, hipComplex mu)
{
    hipComplex alpha = TocuComplex(1.0f); hipComplex beta = TocuComplex(0.0f); //scala = 1/ scala;
    size_t matrixCount = M*M*sizeof(hipComplex);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipComplex *ggtmp;
    hipMalloc((void**)&ggtmp, matrixCount);
    

    for(int i=0;i<s;++i)
    {
        //  matrix_tmp = matrix_tmp @ A 
        cublasAPI<hipComplex>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_P,M,d_P,M,&beta,ggtmp,M);
        checkCudaError(hipMemcpy(d_P,ggtmp,matrixCount,hipMemcpyDeviceToDevice));
    }
    // std::complex<float> exp_mu = std::exp(mu);
    // hipComplex alpha_add = make_hipComplex(exp_mu.real(),exp_mu.imag());
    hipComplex alpha_add = cuCexp(mu);
    checkCublasStatus(cublasAPI<hipComplex>::Scal(cublasH, M*M, &alpha_add,d_P,1));
    checkCudaError(hipFree(ggtmp));
    checkCudaError(hipMemcpy(B,d_P,matrixCount,hipMemcpyDeviceToHost));
    checkCublasStatus(hipblasDestroy(cublasH));
}

void undo_preprocessing(hipDoubleComplex *d_P,hipDoubleComplex *B, int M ,int s, hipDoubleComplex mu)
{
    hipDoubleComplex alpha = TocuDoubleComplex(1.0f); hipDoubleComplex beta = TocuDoubleComplex(0.0f); //scala = 1/ scala;
    size_t matrixCount = M*M*sizeof(hipDoubleComplex);
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    hipDoubleComplex *ggtmp;
    hipMalloc((void**)&ggtmp, matrixCount);
    

    for(int i=0;i<s;++i)
    {
        //  matrix_tmp = matrix_tmp @ A 
        cublasAPI<hipDoubleComplex>::Gemm(cublasH,HIPBLAS_OP_N, HIPBLAS_OP_N, M,M,M, &alpha,d_P,M,d_P,M,&beta,ggtmp,M);
        checkCudaError(hipMemcpy(d_P,ggtmp,matrixCount,hipMemcpyDeviceToDevice));
    }
    // std::complex<double> exp_mu = std::exp(mu);
    // hipDoubleComplex alpha_add = make_hipDoubleComplex(exp_mu.real(),exp_mu.imag());
    hipDoubleComplex alpha_add = cuCexp(mu);
    checkCublasStatus(cublasAPI<hipDoubleComplex>::Scal(cublasH, M*M, &alpha_add,d_P,1));
    checkCudaError(hipFree(ggtmp));
    checkCudaError(hipMemcpy(B,d_P,matrixCount,hipMemcpyDeviceToHost));
    checkCublasStatus(hipblasDestroy(cublasH));
}

float* expm(float *A,const int M)
{
    float *B, *d_A;
    size_t matrixCount = M*M*sizeof(float);
    checkCudaError(hipMalloc((void**)&d_A, matrixCount));
    std::pair<float, int> result = pre_processing(A,d_A,M); // d_A为被预处理后的数组指针
    int s; float mu;
    if (&result.first != nullptr) 
    {
        mu = result.first;
        s = result.second;
    }
    // pade 函数会在里面free d_A的显存
    float *d_P = pade_appromixmate(d_A,M); // 返回pade近似的结果 存储在d_P上，pade_appromixmate返回的是一个已经开辟好内存空间的数组指针，所以直接定义一个数值指针接受就好了，不需要额外开辟空间
    B = (float *)malloc(matrixCount); // B是最后返回的数组指针
    undo_preprocessing(d_P,B,M,s,-mu);
    checkCudaError(hipFree(d_P));
    hipDeviceSynchronize();
    // checkCudaError(hipDeviceReset()); // 这一步是重置GPU设备，作为函数调用时，应该注释这一句
    return B;
}

double* expm(double *A,const int M)
{
    double *B, *d_A;
    size_t matrixCount = M*M*sizeof(double);
    checkCudaError(hipMalloc((void**)&d_A, matrixCount));
    std::pair<double, int> result = pre_processing(A,d_A,M); // d_A为被预处理后的数组指针
    int s; double mu;
    if (&result.first != nullptr) 
    {
        mu = result.first;
        s = result.second;
    }
    // pade 函数会在里面free d_A的显存
    double *d_P = pade_appromixmate(d_A,M); // 返回pade近似的结果 存储在d_P上，pade_appromixmate返回的是一个已经开辟好内存空间的数组指针，所以直接定义一个数值指针接受就好了，不需要额外开辟空间
    B = (double *)malloc(matrixCount); // B是最后返回的数组指针
    undo_preprocessing(d_P,B,M,s,-mu);
    checkCudaError(hipFree(d_P));
    hipDeviceSynchronize();
    // checkCudaError(hipDeviceReset()); // 这一步是重置GPU设备，作为函数调用时，应该注释这一句
    return B;
}

std::complex<float>* expm(std::complex<float> *A,const int M)
{
    hipComplex *d_A;
    size_t matrixCount = M*M*sizeof(std::complex<float>);
    checkCudaError(hipMalloc((void**)&d_A, matrixCount));
    std::pair<hipComplex, int> result = pre_processing(A,d_A,M); // d_A为被预处理后的数组指针
    int s; hipComplex mu;
    if (&result.first != nullptr) 
    {
        mu = result.first;
        s = result.second;
    }
    mu.x = -mu.x;
    mu.y = -mu.y;
    // pade 函数会在里面free d_A的显存
    hipComplex *d_P = pade_appromixmate(d_A,M); // 返回pade近似的结果 存储在d_P上，pade_appromixmate返回的是一个已经开辟好内存空间的数组指针，所以直接定义一个数值指针接受就好了，不需要额外开辟空间
    
    hipComplex *B1;
    B1 = (hipComplex *)malloc(matrixCount); // B是最后返回的数组指针
    undo_preprocessing(d_P,B1,M,s,mu);
    checkCudaError(hipFree(d_P));
    hipDeviceSynchronize();
    std::complex<float> *B = (std::complex<float>*)malloc(matrixCount);
    CuComplexToStdComplex_Array(B1,B,M);
    // checkCudaError(hipDeviceReset()); // 这一步是重置GPU设备，作为函数调用时，应该注释这一句
    return B;
}


std::complex<double>* expm(std::complex<double> *A,const int M)
{
    
    hipDoubleComplex *d_A;
    size_t matrixCount = M*M*sizeof(std::complex<double>);
    checkCudaError(hipMalloc((void**)&d_A, matrixCount));
    std::pair<hipDoubleComplex, int> result = pre_processing(A,d_A,M); // d_A为被预处理后的数组指针
    hipDoubleComplex *T = (hipDoubleComplex *)malloc(matrixCount);
    checkCudaError(hipMemcpy(T,d_A,matrixCount,hipMemcpyDeviceToHost));
    int s; hipDoubleComplex mu;
    if (&result.first != nullptr) 
    {
        mu = result.first;
        s = result.second;
    }
    mu.x = -mu.x;
    mu.y = -mu.y;
    // pade 函数会在里面free d_A的显存
    hipDoubleComplex *d_P = pade_appromixmate(d_A,M); // 返回pade近似的结果 存储在d_P上，pade_appromixmate返回的是一个已经开辟好内存空间的数组指针，所以直接定义一个数值指针接受就好了，不需要额外开辟空间
    hipDoubleComplex *B1;
    B1 = (hipDoubleComplex *)malloc(matrixCount); // B是最后返回的数组指针
    undo_preprocessing(d_P,B1,M,s,mu);
    checkCudaError(hipFree(d_P));
    hipDeviceSynchronize();
    std::complex<double> *B = (std::complex<double>*)malloc(matrixCount);
    CuComplexToStdComplex_Array(B1,B,M);
    // checkCudaError(hipDeviceReset()); // 这一步是重置GPU设备，作为函数调用时，应该注释这一句
    return B;
}